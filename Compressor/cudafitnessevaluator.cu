#include "cudafitnessevaluator.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cstdlib>

using namespace std;
using namespace lossycompressor;

static void HandleError(hipError_t error, const char *file, int line) {
	if (error != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
		scanf(" ");
		exit(EXIT_FAILURE);
	}
}

#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

CudaFitnessEvaluator::CudaFitnessEvaluator(
	int sourceWidth, int sourceHeight,
	int diagramPointsCount, 
	uint8_t * sourceImageData, int sourceDataRowWidthInBytes)
	: FitnessEvaluator(sourceWidth, sourceHeight, diagramPointsCount, sourceImageData, sourceDataRowWidthInBytes) {

	CHECK_ERROR(hipMalloc((void**)&rSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&rCounts, diagramPointsCount*sizeof(int)));
	CHECK_ERROR(hipMalloc((void**)&gSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&gCounts, diagramPointsCount*sizeof(int)));
	CHECK_ERROR(hipMalloc((void**)&bSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&bCounts, diagramPointsCount*sizeof(int)));
	
	CHECK_ERROR(hipMalloc((void**)&colorsTmp, diagramPointsCount*sizeof(Color24bit)));
	CHECK_ERROR(hipMalloc((void**)&pixelPointAssignment, sourceHeight*sizeof(int*)));
	// TODO Maybe make pixel point assignment one dimensional
}

CudaFitnessEvaluator::~CudaFitnessEvaluator() {}



float CudaFitnessEvaluator::calculateFitnessInternal(VoronoiDiagram * diagram) {
	float * devFitness;
	CHECK_ERROR(hipMalloc((void**)&devFitness, sizeof(float)));
	VoronoiDiagram * devDiagram;
	CHECK_ERROR(hipMalloc((void**)&devDiagram, sizeof(VoronoiDiagram)));
	CHECK_ERROR(hipMemcpy(devDiagram, diagram, sizeof(VoronoiDiagram), hipMemcpyHostToDevice));



	// Copy back result fitness
	float fitness;
	CHECK_ERROR(hipMemcpy(&fitness, devFitness, sizeof(float), hipMemcpyDeviceToHost));

	CHECK_ERROR(hipFree(devDiagram));
	return fitness;
}