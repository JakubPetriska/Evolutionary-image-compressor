#include "cudafitnessevaluator.h"
#include "hip/hip_runtime.h"
#include ""

using namespace lossycompressor;

CudaFitnessEvaluator::CudaFitnessEvaluator(
	int sourceWidth, int sourceHeight,
	int diagramPointsCount, uint8_t ** sourceImageData)
	: FitnessEvaluator(sourceWidth, sourceHeight, diagramPointsCount, sourceImageData) {

}

CudaFitnessEvaluator::~CudaFitnessEvaluator() {}

float CudaFitnessEvaluator::calculateFitnessInternal(VoronoiDiagram * diagram) {
	return 5;
}