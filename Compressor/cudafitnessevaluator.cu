#include "hip/hip_runtime.h"
#include "cudafitnessevaluator.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cstdlib>

using namespace std;
using namespace lossycompressor;

#define BLOCK_SIZE 32

static void HandleError(hipError_t error, const char *file, int line) {
	if (error != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
		scanf(" ");
		exit(EXIT_FAILURE);
	}
}

#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

CudaFitnessEvaluator::CudaFitnessEvaluator(
	int sourceWidth, int sourceHeight,
	int diagramPointsCount,
	uint8_t * sourceImageData, int sourceDataRowWidthInBytes)
	: FitnessEvaluator(sourceWidth, sourceHeight, diagramPointsCount, sourceImageData, sourceDataRowWidthInBytes) {

	CHECK_ERROR(hipMalloc((void**)&rSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&rCounts, diagramPointsCount*sizeof(int)));
	CHECK_ERROR(hipMalloc((void**)&gSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&gCounts, diagramPointsCount*sizeof(int)));
	CHECK_ERROR(hipMalloc((void**)&bSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&bCounts, diagramPointsCount*sizeof(int)));

	CHECK_ERROR(hipMalloc((void**)&colorsTmp, diagramPointsCount*sizeof(Color24bit)));
	CHECK_ERROR(hipMalloc((void**)&pixelPointAssignment, sourceHeight*sourceWidth*sizeof(int)));

	int sourceDataSize = sourceHeight*sourceWidth * 3 * sizeof(uint8_t);
	CHECK_ERROR(hipMalloc((void**)&devSourceImageData, sourceDataSize));
	CHECK_ERROR(hipMemcpy(devSourceImageData, sourceImageData, sourceDataSize, hipMemcpyHostToDevice));
}

CudaFitnessEvaluator::~CudaFitnessEvaluator() {}

//__device__ int calculateDiagramPointIndexForPixel(VoronoiDiagram * diagram,
//	int pixelXCoord, int pixelYCoord) {
//
//	int startIndex = findClosestHorizontalPoint(diagram, pixelXCoord, pixelYCoord);
//	int currentClosestPointIndex = startIndex;
//	double squareDistanceToClosest = Utils::calculateSquareDistance(
//		diagram->x(currentClosestPointIndex), diagram->y(currentClosestPointIndex),
//		pixelXCoord, pixelYCoord);
//	bool unacceptableLowerFound = false;
//	bool unacceptableHigherFound = false;
//
//	bool lower = false;
//	for (int i = 1; i <= diagramPointsCount; i = lower ? i : i + 1) {
//		if (unacceptableLowerFound && unacceptableHigherFound) {
//			break;
//		}
//
//		lower = !lower;
//		if ((lower && unacceptableLowerFound)
//			|| (!lower && unacceptableHigherFound)) {
//			continue;
//		}
//
//		int currentIndex = lower ? startIndex - i : startIndex + i;
//		if (currentIndex < 0 || currentIndex >= diagramPointsCount) {
//			if (lower) {
//				unacceptableLowerFound = true;
//			}
//			else {
//				unacceptableHigherFound = true;
//			}
//			continue;
//		}
//
//		double squareDistanceToCurrent = Utils::calculateSquareDistance(
//			diagram->x(currentIndex), diagram->y(currentIndex),
//			pixelXCoord, pixelYCoord);
//
//		if (squareDistanceToCurrent < squareDistanceToClosest) {
//			currentClosestPointIndex = currentIndex;
//			squareDistanceToClosest = squareDistanceToCurrent;
//			unacceptableLowerFound = false;
//			unacceptableHigherFound = false;
//		}
//		else if (lower && !unacceptableLowerFound && diagram->x(currentIndex) < pixelXCoord
//			&& pow(diagram->x(currentIndex) - pixelXCoord, 2) > squareDistanceToClosest) {
//			unacceptableLowerFound = true;
//		}
//		else if (!lower && !unacceptableHigherFound && diagram->x(currentIndex) > pixelXCoord
//			&& pow(diagram->x(currentIndex) - pixelXCoord, 2) > squareDistanceToClosest) {
//			unacceptableHigherFound = true;
//		}
//	}
//
//	return currentClosestPointIndex;
//}
//
//__device__ int findClosestHorizontalPoint(VoronoiDiagram * diagram, int32_t pixelX, int32_t pixelY) {
//	if (diagramPointsCount == 1) {
//		return 0;
//	}
//
//	int start = 0, end = diagramPointsCount;
//	while (start < end - 2) {
//		int pivotIndex = (start + end) / 2;
//		int pixelPivotComparison = CompressorUtils::compare(pixelX, pixelY, diagram->x(pivotIndex), diagram->y(pivotIndex));
//		if (pixelPivotComparison == 0) {
//			return pivotIndex;
//		}
//		else if (pixelPivotComparison < 0) {
//			end = pivotIndex + 1;
//		}
//		else {
//			start = pivotIndex;
//		}
//	}
//
//	assert(start == end - 2);
//
//	double startPixelSquareDist = Utils::calculateSquareDistance(pixelX, pixelY, diagram->x(start), diagram->y(start));
//	double endPixelSquareDist = Utils::calculateSquareDistance(pixelX, pixelY, diagram->x(end), diagram->y(end));
//	if (startPixelSquareDist < endPixelSquareDist) {
//		return start;
//	}
//	else {
//		return end;
//	}
//}

__global__ void fitnessKernel(
	VoronoiDiagram * devDiagram,
	float * outputFitness,
	int diagramPointsCount,
	int sourceWidth,
	int sourceHeight,
	uint8_t * devSourceImageData,
	int sourceDataRowWidthInBytes,
	float * rSums,
	int * rCounts,
	float * gSums,
	int * gCounts,
	float * bSums,
	int * bCounts,
	Color24bit * colorsTmp,
	int * pixelPointAssignment) {

	int pixelHorizontal = blockIdx.x * blockDim.x + threadIdx.x;
	int pixelVertical = blockIdx.y * blockDim.y + threadIdx.y;
	int linearIndex = pixelHorizontal * sourceWidth + pixelVertical;

	// If pixel of this thread is in images
	if (pixelHorizontal < sourceWidth && pixelVertical < sourceHeight) {
		// Reset the work variables
		if (linearIndex < diagramPointsCount) {
			rSums[linearIndex] = 0;
			rCounts[linearIndex] = 0;
			gSums[linearIndex] = 0;
			gCounts[linearIndex] = 0;
			bSums[linearIndex] = 0;
			bCounts[linearIndex] = 0;
		}

		// Find diagram points for all pixels and calculate colors of individual points
		//int pointIndex = calculateDiagramPointIndexForPixel(diagram, j, i);
		int pointIndex = 1;
		
		pixelPointAssignment[linearIndex] = pointIndex;

		int colorStartIndexInSourceData = pixelVertical * sourceDataRowWidthInBytes + pixelHorizontal * 3;
		bSums[pointIndex] += devSourceImageData[colorStartIndexInSourceData];
		bCounts[pointIndex] += 1;
		gSums[pointIndex] += devSourceImageData[colorStartIndexInSourceData + 1];
		gCounts[pointIndex] += 1;
		rSums[pointIndex] += devSourceImageData[colorStartIndexInSourceData + 2];
		rCounts[pointIndex] += 1;


		// Calculate the fitness of this pixel
		Color24bit color = colorsTmp[pointIndex];

		float pixelDeviation
			= (fabsf((float)(devSourceImageData[colorStartIndexInSourceData] - color.b)) // Absolute red color deviation
			+ fabsf((float)(devSourceImageData[colorStartIndexInSourceData + 1] - color.g)) // Absolute green color deviation
			+ fabsf((float)(devSourceImageData[colorStartIndexInSourceData + 2] - color.r))) // Absolute blue color deviation
			/ 255.0f;

		atomicAdd(outputFitness, pixelDeviation);
	}
}

float CudaFitnessEvaluator::calculateFitnessInternal(VoronoiDiagram * diagram) {
	float * devFitness;
	CHECK_ERROR(hipMalloc((void**)&devFitness, sizeof(float)));
	VoronoiDiagram * devDiagram;
	CHECK_ERROR(hipMalloc((void**)&devDiagram, sizeof(VoronoiDiagram)));
	CHECK_ERROR(hipMemcpy(devDiagram, diagram, sizeof(VoronoiDiagram), hipMemcpyHostToDevice));

	int gridWidth = sourceWidth / BLOCK_SIZE;
	if (sourceWidth - gridWidth * BLOCK_SIZE > 0) {
		++gridWidth;
	}
	int gridHeight = sourceHeight / BLOCK_SIZE;
	if (sourceHeight - gridHeight * BLOCK_SIZE > 0) {
		++gridHeight;
	}

	dim3 blocks(gridWidth, gridHeight);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	fitnessKernel << <blocks, threads >> >(
		devDiagram, devFitness,
		diagramPointsCount, sourceWidth, sourceHeight,
		devSourceImageData, sourceDataRowWidthInBytes,
		rSums, rCounts, gSums, gCounts, bSums, bCounts,
		colorsTmp, pixelPointAssignment);

	// Copy back result fitness
	float fitness;
	CHECK_ERROR(hipMemcpy(&fitness, devFitness, sizeof(float), hipMemcpyDeviceToHost));

	CHECK_ERROR(hipFree(devDiagram));
	return fitness;
}