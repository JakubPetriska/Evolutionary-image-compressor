#include "hip/hip_runtime.h"
#include "cudafitnessevaluator.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cstdlib>
#include "compressorutils.h"
#include "utils.h"

using namespace std;
using namespace lossycompressor;

#define BLOCK_SIZE 32

static void HandleError(hipError_t error, const char *file, int line) {
	if (error != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
		exit(EXIT_FAILURE);
	}
}

#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

CudaFitnessEvaluator::CudaFitnessEvaluator(
	int sourceWidth, int sourceHeight,
	int diagramPointsCount,
	uint8_t * sourceImageData, int sourceDataRowWidthInBytes)
	: FitnessEvaluator(sourceWidth, sourceHeight, diagramPointsCount, sourceImageData, sourceDataRowWidthInBytes) {

	CHECK_ERROR(hipMalloc((void**)&rSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&rCounts, diagramPointsCount*sizeof(int)));
	CHECK_ERROR(hipMalloc((void**)&gSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&gCounts, diagramPointsCount*sizeof(int)));
	CHECK_ERROR(hipMalloc((void**)&bSums, diagramPointsCount*sizeof(float)));
	CHECK_ERROR(hipMalloc((void**)&bCounts, diagramPointsCount*sizeof(int)));

	CHECK_ERROR(hipMalloc((void**)&colors, diagramPointsCount*sizeof(Color24bit)));
	CHECK_ERROR(hipMalloc((void**)&pixelPointAssignment, sourceHeight*sourceWidth*sizeof(int)));

	int sourceDataSize = sourceHeight*sourceWidth * 3 * sizeof(uint8_t);
	CHECK_ERROR(hipMalloc((void**)&devSourceImageData, sourceDataSize));
	CHECK_ERROR(hipMemcpy(devSourceImageData, sourceImageData, sourceDataSize, hipMemcpyHostToDevice));

	// Allocate arrays for voronoi diagram saved on device
	diagramPointsCoordinatesSize = diagramPointsCount * sizeof(int32_t);
	int32_t * devDiagramPointsXCoordinates;
	CHECK_ERROR(hipMalloc((void**)&devDiagramPointsXCoordinates, diagramPointsCoordinatesSize));
	int32_t * devDiagramPointsYCoordinates;
	CHECK_ERROR(hipMalloc((void**)&devDiagramPointsYCoordinates, diagramPointsCoordinatesSize));

	diagram = new VoronoiDiagram(diagramPointsCount, devDiagramPointsXCoordinates, devDiagramPointsYCoordinates);
	CHECK_ERROR(hipMalloc((void**)&devDiagram, sizeof(VoronoiDiagram)));
	CHECK_ERROR(hipMemcpy(devDiagram, diagram, sizeof(VoronoiDiagram), hipMemcpyHostToDevice));

	// TODO release all these
}

CudaFitnessEvaluator::~CudaFitnessEvaluator() {}

__device__ int compare(int firstX, int firstY, int secondX, int secondY) {
	if (firstX == secondX && firstY == secondY) {
		return 0;
	}
	else if (firstX < secondX
		|| (firstX == secondX && firstY < secondY)) {
		return -1;
	}
	else {
		return 1;
	}
}

__device__ int32_t x(VoronoiDiagram * diagram, int index) {
	return diagram->diagramPointsXCoordinates[index];
}

__device__ int32_t y(VoronoiDiagram * diagram, int index) {
	return diagram->diagramPointsYCoordinates[index];
}

__device__ double calculateSquareDistance(int firstX, int firstY, int secondX, int secondY) {
	return ((firstX - secondX) * (firstX - secondX)) + ((firstY - secondY) * (firstY - secondY));
}

__device__ int findClosestHorizontalPoint(int diagramPointsCount, VoronoiDiagram * diagram, int pixelX, int pixelY) {
	if (diagramPointsCount == 1) {
		return 0;
	}

	int start = 0, end = diagramPointsCount;
	while (start < end - 2) {
		int pivotIndex = (start + end) / 2;
		int pixelPivotComparison = compare(pixelX, pixelY, x(diagram, pivotIndex), y(diagram, pivotIndex));
		if (pixelPivotComparison == 0) {
			return pivotIndex;
		}
		else if (pixelPivotComparison < 0) {
			end = pivotIndex + 1;
		}
		else {
			start = pivotIndex;
		}
	}

	double startPixelSquareDist = calculateSquareDistance(pixelX, pixelY, x(diagram, start), y(diagram, start));
	double endPixelSquareDist = calculateSquareDistance(pixelX, pixelY, x(diagram, end), y(diagram, end));
	if (startPixelSquareDist < endPixelSquareDist) {
		return start;
	}
	else {
		return end;
	}
}

__device__ int calculateDiagramPointIndexForPixel(int diagramPointsCount, VoronoiDiagram * diagram,
	int pixelXCoord, int pixelYCoord) {

	int startIndex = findClosestHorizontalPoint(diagramPointsCount, diagram, pixelXCoord, pixelYCoord);
	int currentClosestPointIndex = startIndex;
	double squareDistanceToClosest = calculateSquareDistance(
		x(diagram, currentClosestPointIndex), y(diagram, currentClosestPointIndex),
		pixelXCoord, pixelYCoord);

	bool unacceptableLowerFound = false;
	bool unacceptableHigherFound = false;

	bool lower = false;
	for (int i = 1; i <= diagramPointsCount; i = lower ? i : i + 1) {
		if (unacceptableLowerFound && unacceptableHigherFound) {
			break;
		}

		lower = !lower;
		if ((lower && unacceptableLowerFound)
			|| (!lower && unacceptableHigherFound)) {
			continue;
		}

		int currentIndex = lower ? startIndex - i : startIndex + i;
		if (currentIndex < 0 || currentIndex >= diagramPointsCount) {
			if (lower) {
				unacceptableLowerFound = true;
			}
			else {
				unacceptableHigherFound = true;
			}
			continue;
		}

		double squareDistanceToCurrent = calculateSquareDistance(
			x(diagram, currentIndex), y(diagram, currentIndex),
			pixelXCoord, pixelYCoord);

		if (squareDistanceToCurrent < squareDistanceToClosest) {
			currentClosestPointIndex = currentIndex;
			squareDistanceToClosest = squareDistanceToCurrent;
			unacceptableLowerFound = false;
			unacceptableHigherFound = false;
		}
		else if (lower && !unacceptableLowerFound && x(diagram, currentIndex) < pixelXCoord
			&& ((x(diagram, currentIndex) - pixelXCoord) * (x(diagram, currentIndex) - pixelXCoord)) > squareDistanceToClosest) {
			unacceptableLowerFound = true;
		}
		else if (!lower && !unacceptableHigherFound && x(diagram, currentIndex) > pixelXCoord
			&& ((x(diagram, currentIndex) - pixelXCoord) * (x(diagram, currentIndex) - pixelXCoord)) > squareDistanceToClosest) {
			unacceptableHigherFound = true;
		}
	}

	return currentClosestPointIndex;
}

__global__ void resetWorkVarsKernel(
	int diagramPointsCount,
	int sourceWidth,
	int sourceHeight,
	float * rSums,
	int * rCounts,
	float * gSums,
	int * gCounts,
	float * bSums,
	int * bCounts) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < diagramPointsCount) {
		rSums[index] = 0;
		rCounts[index] = 0;
		gSums[index] = 0;
		gCounts[index] = 0;
		bSums[index] = 0;
		bCounts[index] = 0;
	}
}

__global__ void calculateColorsSumsKernel(
	VoronoiDiagram * devDiagram,
	int diagramPointsCount,
	int sourceWidth,
	int sourceHeight,
	uint8_t * devSourceImageData,
	int sourceDataRowWidthInBytes,
	float * rSums,
	int * rCounts,
	float * gSums,
	int * gCounts,
	float * bSums,
	int * bCounts,
	int * pixelPointAssignment) {

	int pixelHorizontal = blockIdx.x * blockDim.x + threadIdx.x;
	int pixelVertical = blockIdx.y * blockDim.y + threadIdx.y;

	// If pixel of this thread is in the image
	if (pixelHorizontal < sourceWidth && pixelVertical < sourceHeight) {
		int linearIndex = pixelHorizontal + sourceWidth * pixelVertical;
		int colorStartIndexInSourceData = pixelHorizontal * 3 + pixelVertical * sourceDataRowWidthInBytes;

		// Find diagram points for all pixels and calculate colors of individual points
		int pointIndex = calculateDiagramPointIndexForPixel(diagramPointsCount, devDiagram, pixelHorizontal, pixelVertical);

		pixelPointAssignment[linearIndex] = pointIndex;

		atomicAdd(bSums + pointIndex, devSourceImageData[colorStartIndexInSourceData]);
		atomicAdd(bCounts + pointIndex, 1);
		atomicAdd(gSums + pointIndex, devSourceImageData[colorStartIndexInSourceData + 1]);
		atomicAdd(gCounts + pointIndex, 1);
		atomicAdd(rSums + pointIndex, devSourceImageData[colorStartIndexInSourceData + 2]);
		atomicAdd(rCounts + pointIndex, 1);
	}
}

__global__ void calculateColorsKernel(
	int diagramPointsCount,
	int sourceWidth,
	int sourceHeight,
	float * rSums,
	int * rCounts,
	float * gSums,
	int * gCounts,
	float * bSums,
	int * bCounts,
	Color24bit * colors) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < diagramPointsCount) {
		Color24bit * color = &colors[index];
		color->b = (uint8_t)(bSums[index] / bCounts[index] + 0.5);
		color->g = (uint8_t)(gSums[index] / gCounts[index] + 0.5);
		color->r = (uint8_t)(rSums[index] / rCounts[index] + 0.5);
	}
}

__global__ void calculateFitnessKernel(
	float * outputFitness,
	int sourceWidth,
	int sourceHeight,
	uint8_t * devSourceImageData,
	int sourceDataRowWidthInBytes,
	Color24bit * colors,
	int * pixelPointAssignment) {

	int pixelHorizontal = blockIdx.x * blockDim.x + threadIdx.x;
	int pixelVertical = blockIdx.y * blockDim.y + threadIdx.y;

	// If pixel of this thread is in images
	if (pixelHorizontal < sourceWidth && pixelVertical < sourceHeight) {
		int linearIndex = pixelHorizontal + pixelVertical * sourceWidth;
		int colorStartIndexInSourceData = pixelVertical * sourceDataRowWidthInBytes + pixelHorizontal * 3;

		int pointIndex = pixelPointAssignment[linearIndex];
		Color24bit color = colors[pointIndex];

		float pixelDeviation
			= (fabsf((float)(devSourceImageData[colorStartIndexInSourceData] - color.b)) // Absolute red color deviation
			+ fabsf((float)(devSourceImageData[colorStartIndexInSourceData + 1] - color.g)) // Absolute green color deviation
			+ fabsf((float)(devSourceImageData[colorStartIndexInSourceData + 2] - color.r))); // Absolute blue color deviation

		atomicAdd(outputFitness, pixelDeviation);
	}
}

float CudaFitnessEvaluator::calculateFitnessInternal(VoronoiDiagram * diagram) {
	float * devFitness;
	CHECK_ERROR(hipMalloc((void**)&devFitness, sizeof(float)));
	CHECK_ERROR(hipMemset((void*)devFitness, 0, sizeof(float)));

	CHECK_ERROR(hipMemcpy(this->diagram->diagramPointsXCoordinates, diagram->diagramPointsXCoordinates,
		diagramPointsCoordinatesSize, hipMemcpyHostToDevice));
	CHECK_ERROR(hipMemcpy(this->diagram->diagramPointsYCoordinates, diagram->diagramPointsYCoordinates,
		diagramPointsCoordinatesSize, hipMemcpyHostToDevice));

	int everyPointThreadCount = BLOCK_SIZE * BLOCK_SIZE;
	int everyPointBlocksCount = diagramPointsCount / everyPointThreadCount;
	if (diagramPointsCount - everyPointBlocksCount * everyPointThreadCount > 0) {
		++everyPointBlocksCount;
	}

	int gridWidth = sourceWidth / BLOCK_SIZE;
	if (sourceWidth - gridWidth * BLOCK_SIZE > 0) {
		++gridWidth;
	}
	int gridHeight = sourceHeight / BLOCK_SIZE;
	if (sourceHeight - gridHeight * BLOCK_SIZE > 0) {
		++gridHeight;
	}

	dim3 everyPixelBlocks(gridWidth, gridHeight);
	dim3 everyPixelThreads(BLOCK_SIZE, BLOCK_SIZE);

	resetWorkVarsKernel << <everyPointBlocksCount, everyPointThreadCount >> >(
		diagramPointsCount,
		sourceWidth, sourceHeight,
		rSums, rCounts, gSums, gCounts, bSums, bCounts);

	calculateColorsSumsKernel << <everyPixelBlocks, everyPixelThreads >> >(
		devDiagram, diagramPointsCount,
		sourceWidth, sourceHeight,
		devSourceImageData, sourceDataRowWidthInBytes,
		rSums, rCounts, gSums, gCounts, bSums, bCounts,
		pixelPointAssignment);

	calculateColorsKernel << <everyPointBlocksCount, everyPointThreadCount >> >(
		diagramPointsCount,
		sourceWidth, sourceHeight,
		rSums, rCounts, gSums, gCounts, bSums, bCounts,
		colors);

	calculateFitnessKernel << <everyPixelBlocks, everyPixelThreads >> >(
		devFitness,
		sourceWidth, sourceHeight,
		devSourceImageData,
		sourceDataRowWidthInBytes,
		colors, pixelPointAssignment);

	// Copy back result fitness
	float fitness = 0;
	CHECK_ERROR(hipMemcpy(&fitness, devFitness, sizeof(float), hipMemcpyDeviceToHost));
	return fitness / (sourceWidth * sourceHeight);
}